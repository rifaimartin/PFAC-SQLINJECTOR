#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <cctype>
#include <unordered_map>
#include <queue>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <regex>

using namespace std;

#define ALPHABET_SIZE 128
#define MAX_NODES     8192    // adjust as needed
#define MAX_PATTERNS  1024     // adjust as needed
#define THREADS_PER_BLOCK 256

// Sparse transition representation
struct SparseTransition {
    unsigned char character;
    int next_state;
};

// Macro for CUDA error checking
#define CUDA_CHECK(call)                                                         \
    do {                                                                         \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            cerr << "CUDA error at " << __FILE__ << ":" << __LINE__     \
                      << ": " << hipGetErrorString(err) << endl;        \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    } while (0)

// Device memory for sparse transition table - MOVED TO GLOBAL MEMORY INSTEAD OF CONSTANT
__device__ __constant__ int d_pattern_weights[MAX_PATTERNS];    // Keep this in constant memory

// Optimized PFAC kernel - one thread per query
__global__ void PFACKernel(
    const char* __restrict__ d_queries,
    const int*  __restrict__ d_offsets,
    const int*  __restrict__ d_lengths,
    int          numQueries,
    int*        __restrict__ d_risk_scores,
    // Pass tables as kernel parameters instead of using constant memory
    const int* __restrict__ d_transition_offsets,
    const SparseTransition* __restrict__ d_transitions,
    const uint64_t* __restrict__ d_match_masks)
{
    int query_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (query_idx >= numQueries) return;
    
    // Get query details
    int offset = d_offsets[query_idx];
    int length = d_lengths[query_idx];
    
    // Thread-local variables to store matched patterns
    // Using register memory for highest performance
    uint64_t matched_low = 0;
    uint64_t matched_high = 0;
    int risk_score = 0;
    
    // Process each starting position
    for (int start_pos = 0; start_pos < length; ++start_pos) {
        int state = 0;  // Start from root
        
        // Process characters from this position
        for (int i = start_pos; i < length; ++i) {
            unsigned char c = (unsigned char)d_queries[offset + i];
            if (c >= ALPHABET_SIZE) break;
            
            // Find transition using sparse representation
            int next_state = -1;
            int transition_start = d_transition_offsets[state];
            int transition_end = d_transition_offsets[state + 1];
            
            // Linear search through sparse transitions
            for (int t = transition_start; t < transition_end; ++t) {
                if (d_transitions[t].character == c) {
                    next_state = d_transitions[t].next_state;
                    break;
                }
            }
            
            // If no transition found, break
            if (next_state == -1) break;
            state = next_state;
            
            // Check for matches at this state
            uint64_t new_matches_low = d_match_masks[state * 2] & ~matched_low;
            uint64_t new_matches_high = d_match_masks[state * 2 + 1] & ~matched_high;
            
            // Process new low matches
            while (new_matches_low) {
                int pattern_id = __ffsll(new_matches_low) - 1;
                risk_score += d_pattern_weights[pattern_id];
                new_matches_low &= (new_matches_low - 1); // Clear lowest bit
            }
            
            // Process new high matches
            while (new_matches_high) {
                int pattern_id = __ffsll(new_matches_high) - 1 + 64;
                risk_score += d_pattern_weights[pattern_id];
                new_matches_high &= (new_matches_high - 1); // Clear lowest bit
            }
            
            // Update match masks
            matched_low |= d_match_masks[state * 2];
            matched_high |= d_match_masks[state * 2 + 1];
        }
        
        // Reset match tracking for next starting position
        matched_low = 0;
        matched_high = 0;
    }
    
    // Write final risk score for this query
    d_risk_scores[query_idx] = risk_score;
}

// CPU Trie Node for building automaton
struct TrieNode {
    unordered_map<char, TrieNode*> children;
    vector<int> pattern_ids;
    int id;
    
    TrieNode(int id): id(id) {}
};

class PFACTrie {
public:
    PFACTrie() { 
        root = new TrieNode(0); 
        nodes.push_back(root);
    }
    
    ~PFACTrie() {
        for (auto node : nodes) {
            delete node;
        }
    }

    void insert(const string& pat, int pattern_id) {
        TrieNode* current = root;
        
        for (char ch : pat) {
            if (!current->children.count(ch)) {
                TrieNode* newNode = new TrieNode(nodes.size());
                current->children[ch] = newNode;
                nodes.push_back(newNode);
            }
            current = current->children[ch];
        }
        
        current->pattern_ids.push_back(pattern_id);
    }
    
    // Build sparse transition tables
    void buildSparseTransitionTables(
        vector<int>& transition_offsets,
        vector<SparseTransition>& transitions,
        vector<uint64_t>& match_masks)
    {
        int numNodes = nodes.size();
        transition_offsets.resize(numNodes + 1);
        match_masks.resize(numNodes * 2, 0); // 2 uint64_t per node
        
        // Count transitions per node
        int totalTransitions = 0;
        for (int i = 0; i < numNodes; i++) {
            transition_offsets[i] = totalTransitions;
            totalTransitions += nodes[i]->children.size();
        }
        transition_offsets[numNodes] = totalTransitions;
        
        // Allocate transitions
        transitions.resize(totalTransitions);
        
        // Fill transitions and match masks
        for (int i = 0; i < numNodes; i++) {
            TrieNode* node = nodes[i];
            int transIdx = transition_offsets[i];
            
            // Add transitions for this node
            for (auto& kv : node->children) {
                unsigned char ch = kv.first;
                TrieNode* child = kv.second;
                transitions[transIdx].character = ch;
                transitions[transIdx].next_state = child->id;
                transIdx++;
            }
            
            // Set match masks
            for (int pid : node->pattern_ids) {
                if (pid < 64) {
                    match_masks[i * 2] |= (1ULL << pid);
                } else {
                    match_masks[i * 2 + 1] |= (1ULL << (pid - 64));
                }
            }
        }
    }
    
    int nodeCount() const { return nodes.size(); }

private:
    TrieNode* root;
    vector<TrieNode*> nodes;
};

// Text normalization
string normalize(const string &s) {
    string r = s;
    transform(r.begin(), r.end(), r.begin(), ::tolower);
    return r;
}

// Risk classification
string classifyRisk(int score) {
    if (score <= 30)   return "low";
    if (score <= 70)   return "medium";
    if (score <= 90)   return "high";
    return "critical";
}

// Function to read patterns from file
vector<string> readPatternsFromFile(const string& filename) {
    vector<string> patterns;
    ifstream file(filename);
    
    if (!file.is_open()) {
        cerr << "Error: Could not open pattern file: " << filename << endl;
        return patterns;
    }
    
    // Read whole file into a string
    string content((istreambuf_iterator<char>(file)), istreambuf_iterator<char>());
    file.close();
    
    // Process with regex to extract patterns more reliably
    regex patternRegex("\"([^\"]+)\"");
    auto matches_begin = sregex_iterator(content.begin(), content.end(), patternRegex);
    auto matches_end = sregex_iterator();
    
    for (sregex_iterator i = matches_begin; i != matches_end; ++i) {
        smatch match = *i;
        string pattern = match[1].str(); // Get the content inside the quotes
        
        // Remove trailing comma if present
        if (!pattern.empty() && pattern.back() == ',') {
            pattern.pop_back();
        }
        
        // Add to patterns list
        if (!pattern.empty()) {
            patterns.push_back(pattern);
        }
    }
    
    cout << "Extracted " << patterns.size() << " patterns using regex" << endl;
    
    // If no patterns found, try a different approach
    if (patterns.empty()) {
        file.open(filename);
        string line;
        while (getline(file, line)) {
            // Skip empty lines and comment lines
            if (line.empty() || line[0] == '#') continue;
            
            // Remove leading/trailing whitespace
            line.erase(0, line.find_first_not_of(" \t\r\n"));
            line.erase(line.find_last_not_of(" \t\r\n") + 1);
            
            // Remove quotes and comma if present
            if (line.size() >= 2 && line.front() == '"' && 
                (line.back() == '"' || (line.size() >= 3 && line[line.size()-2] == '"' && line.back() == ','))) {
                
                // Remove opening quote
                line = line.substr(1);
                
                // Remove closing quote and optional comma
                if (line.back() == ',') {
                    line = line.substr(0, line.size() - 2);  // Remove both " and ,
                } else if (line.back() == '"') {
                    line = line.substr(0, line.size() - 1);  // Remove just "
                }
            } else if (line.back() == ',') {
                // If only comma present at end with no quotes
                line = line.substr(0, line.size() - 1);
            }
            
            // Add pattern if it's not empty after processing
            if (!line.empty()) {
                patterns.push_back(line);
            }
        }
        file.close();
        cout << "Secondary extraction found " << patterns.size() << " patterns" << endl;
    }
   
    
    return patterns;
}

void printUsage(const char* programName) {
    cout << "Usage: " << programName << " [options]" << endl;
    cout << "Options:" << endl;
    cout << "  -d, --dataset <file>   CSV dataset file to process (default: sql_dataset_Critical_10000.csv)" << endl;
    cout << "  -p, --patterns <file>  Pattern file to use (default: patterns.txt)" << endl;
    cout << "  -o, --output <file>    Output file for results (default: results.txt)" << endl;
    cout << "  -h, --help             Show this help message" << endl;
}

int main(int argc, char** argv) {
    // Default filenames
    string datasetFile = "sql_dataset_Critical_10000";
    // string patternFile = "patterns.txt";
    // string outputFile = "results_pfac.txt";

    for (int i = 1; i < argc; i++)
    {
        string arg = argv[i];

        if (arg == "-h" || arg == "--help")
        {
            printUsage(argv[0]);
            return 0;
        }
        else if (arg == "-d" || arg == "--dataset")
        {
            datasetFile = argv[++i];
        }
        // else if (arg == "-o" || arg == "--output") {
        //     outputFile = argv[++i];
        // }
        else
        {
            cerr << "Unknown option: " << arg << endl;
            printUsage(argv[0]);
            return 1;
        }
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    ofstream out("result_PFAC_" + datasetFile + ".txt");
    streambuf* coutbuf = cout.rdbuf(); 
    cout.rdbuf(out.rdbuf());

    // cout << "=== Optimized Sparse PFAC for SQL Injection Detection ===" << endl;

    // 1. Load patterns
    vector<string> rawPatterns;
    
    string patternFile = "patterns.txt";
    
    ifstream testFile(patternFile);
    if (testFile.is_open()) {
        testFile.close();
        rawPatterns = readPatternsFromFile(patternFile);
        if (!rawPatterns.empty()) {
            cout << "Successfully read " << rawPatterns.size() << " patterns from " << patternFile << endl;
        } else {
            cout << "Warning: Could not extract any patterns from " << patternFile << endl;
        }
    } else {
        cout << "Warning: Could not open pattern file " << patternFile << ". Using default patterns." << endl;
    }

    int P = rawPatterns.size();
    
    // Display first few patterns
    cout << "First " << min(10, P) << " patterns:" << endl;
    for (int i = 0; i < min(10, P); i++) {
        cout << i+1 << ". " << rawPatterns[i] << endl;
    }
    if (P > 10) {
        cout << "... and " << (P-10) << " more patterns" << endl;
    }

    // Normalize patterns and assign weights
    vector<string> patterns(P);
    vector<int> weights(P);
    
    for (int i = 0; i < P; ++i) {
        patterns[i] = normalize(rawPatterns[i]);
        const auto &pat = patterns[i];
        
        // Assign weights based on pattern content
        if (pat.find("; drop") != string::npos || 
            pat.find("xp_cmdshell") != string::npos ||
            pat.find("; exec") != string::npos || 
            pat.find("outfile") != string::npos ||
            pat.find("load_file") != string::npos) {
            weights[i] = 100;
        }
        else if (pat.find("; delete") != string::npos || 
                 pat.find("; insert") != string::npos ||
                 pat.find("; truncate") != string::npos || 
                 pat.find("; update") != string::npos ||
                 pat.find("sleep(") != string::npos || 
                 pat.find("version(") != string::npos ||
                 pat.find("current_user") != string::npos) {
            weights[i] = 15;
        }
        else {
            weights[i] = 10;
        }
    }

    // Print weights statistics
    int weight10 = 0, weight15 = 0, weight100 = 0;
    for (int w : weights) {
        if (w == 10) weight10++;
        else if (w == 15) weight15++;
        else if (w == 100) weight100++;
    }
    cout << "Weight distribution: 10=" << weight10 << ", 15=" << weight15 << ", 100=" << weight100 << endl;
    
    // Build PFAC Trie
    PFACTrie trie;
    for (int i = 0; i < P; ++i) {
        trie.insert(patterns[i], i);
    }
    
    //  Build sparse transition tables
    vector<int> h_transition_offsets;
    vector<SparseTransition> h_transitions;
    vector<uint64_t> h_match_masks;
    trie.buildSparseTransitionTables(h_transition_offsets, h_transitions, h_match_masks);
    
    int N = trie.nodeCount();
    cout << "Trie node count: " << N << endl;
    cout << "Transitions count: " << h_transitions.size() << endl;
    cout << "Average transitions per node: " << (float)h_transitions.size() / N << endl;
    
    // Verify we don't exceed max sizes
    if (N > MAX_NODES) {
        cerr << "Error: Too many nodes in trie (" << N << "), max is " << MAX_NODES << endl;
        return EXIT_FAILURE;
    }
    
    if (P > MAX_PATTERNS) {
        cerr << "Error: Too many patterns (" << P << "), max is " << MAX_PATTERNS << endl;
        return EXIT_FAILURE;
    }
    
    // Allocate GPU memory for transition tables
    int* d_transition_offsets;
    SparseTransition* d_transitions;
    uint64_t* d_match_masks;
    
    CUDA_CHECK(hipMalloc(&d_transition_offsets, (N + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_transitions, h_transitions.size() * sizeof(SparseTransition)));
    CUDA_CHECK(hipMalloc(&d_match_masks, N * 2 * sizeof(uint64_t)));
    
    // Copy transition data to GPU
    CUDA_CHECK(hipMemcpy(d_transition_offsets, h_transition_offsets.data(), 
                        (N + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_transitions, h_transitions.data(), 
                        h_transitions.size() * sizeof(SparseTransition), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_match_masks, h_match_masks.data(), 
                        N * 2 * sizeof(uint64_t), hipMemcpyHostToDevice));
    
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_pattern_weights), weights.data(), 
                                P * sizeof(int)));
    
    ifstream infile(datasetFile+".csv");
    if (!infile.is_open()) {
        cerr << "Error: could not open CSV file.\n";
        return EXIT_FAILURE;
    }
    
    string line;
    vector<string> queries;
    vector<string> expected;
    vector<string> originalQueries; // Keep original for debugging
    
    // Skip header
    getline(infile, line);
    
    while (getline(infile, line)) {
        if (line.empty()) continue;
        stringstream ss(line);
        string q, expRisk, expScore;
        getline(ss, q, ',');
        getline(ss, expRisk, ',');
        getline(ss, expScore, ',');
        
        originalQueries.push_back(q);
        queries.push_back(normalize(q));
        expected.push_back(expRisk);
    }
    infile.close();
    
    int Q = queries.size();
    cout << "Loaded " << Q << " queries." << endl;

    // Prepare query data for GPU
    vector<int> h_offsets(Q), h_lengths(Q);
    int totalLen = 0;
    for (int i = 0; i < Q; ++i) {
        h_offsets[i] = totalLen;
        h_lengths[i] = queries[i].size();
        totalLen += h_lengths[i];
    }
    
    vector<char> h_buffer(totalLen);
    for (int i = 0; i < Q; ++i) {
        memcpy(&h_buffer[h_offsets[i]], queries[i].data(), h_lengths[i]);
    }

    // Allocate GPU memory
    char* d_queries;
    int* d_offsets;
    int* d_lengths;
    int* d_risk_scores;
    
    CUDA_CHECK(hipMalloc(&d_queries, totalLen * sizeof(char)));
    CUDA_CHECK(hipMalloc(&d_offsets, Q * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_lengths, Q * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_risk_scores, Q * sizeof(int)));
    
    CUDA_CHECK(hipMemcpy(d_queries, h_buffer.data(), totalLen * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_offsets, h_offsets.data(), Q * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_lengths, h_lengths.data(), Q * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_risk_scores, 0, Q * sizeof(int)));

    int threadsPerBlock = THREADS_PER_BLOCK;
    int blocks = (Q + threadsPerBlock - 1) / threadsPerBlock;
    
    cout << "Launching kernel with " << blocks << " blocks and " 
         << threadsPerBlock << " threads per block..." << endl;
    
    hipEventRecord(start);
    
    // Launch kernel with one thread per query
    PFACKernel<<<blocks, threadsPerBlock>>>(
        d_queries, d_offsets, d_lengths, Q, d_risk_scores,
        d_transition_offsets, d_transitions, d_match_masks);
    
    hipEventRecord(stop);
    
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    vector<int> h_results(Q);
    CUDA_CHECK(hipMemcpy(h_results.data(), d_risk_scores, Q * sizeof(int), hipMemcpyDeviceToHost));

    int correct = 0;
    int zeroScores = 0;
    
    for (int i = 0; i < Q; ++i) {
        string compRisk = classifyRisk(h_results[i]);
        bool match = (compRisk == expected[i]);
        if (match) ++correct;
        if (h_results[i] == 0) ++zeroScores;
        
        // Print first 10 
        if (i < 10) {
            cout << "Query " << i << ": \"" << originalQueries[i] << "\"" << endl;
            cout << "  Normalized: \"" << queries[i] << "\"" << endl;
            cout << "  Score: " << h_results[i] << ", computed risk: " << compRisk 
                 << ", expected: " << expected[i] << (match ? " [OK]" : " [Mismatch]") << endl;
        }
    }
    
    double accuracy = Q ? (100.0 * correct / Q) : 0.0;
    cout << "\nTotal: " << Q << ", Correct: " << correct
         << ", Accuracy: " << accuracy << "%" << endl;
    cout << "Queries with zero score: " << zeroScores << " (" 
         << (100.0 * zeroScores / Q) << "%)" << endl;
    cout << "Optimized Sparse PFAC kernel execution time: " << milliseconds << " ms" << endl;

    hipFree(d_queries);
    hipFree(d_offsets);
    hipFree(d_lengths);
    hipFree(d_risk_scores);
    hipFree(d_transition_offsets);
    hipFree(d_transitions);
    hipFree(d_match_masks);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);

    cout.rdbuf(coutbuf); 
    return 0;
}