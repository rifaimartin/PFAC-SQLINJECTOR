// aho_cuda_full.cu
// Parallel Aho–Corasick SQLi risk detection using CUDA.

#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <cctype>
#include <unordered_map>
#include <queue>
#include <hip/hip_runtime.h>
#include <stdint.h>

using namespace std;

#define ALPHABET_SIZE 128
#define MAX_NODES     8192    // adjust as needed
#define MAX_PATTERNS  256     // adjust as needed

// Macro for CUDA error checking
#define CUDA_CHECK(call)                                                         \
    do {                                                                         \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            cerr << "CUDA error at " << __FILE__ << ":" << __LINE__     \
                      << ": " << hipGetErrorString(err) << endl;        \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    } while (0)

// Device automaton structures
__device__ int      d_children[MAX_NODES * ALPHABET_SIZE];
__device__ int      d_fail[MAX_NODES];
__device__ uint64_t d_maskLow[MAX_NODES];
__device__ uint64_t d_maskHigh[MAX_NODES];
__device__ int      d_patternWeights[MAX_PATTERNS];

// Kernel: one thread per query
__global__ void ahoSearchKernel(
    const char* __restrict__ d_queries,
    const int*  __restrict__ d_offsets,
    const int*  __restrict__ d_lengths,
    int          numQueries,
    int*        __restrict__ d_results)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= numQueries) return;

    const char* q = d_queries + d_offsets[tid];
    int len = d_lengths[tid];
    int node = 0;
    uint64_t maskL = 0, maskH = 0;
    int risk = 0;

    for (int i = 0; i < len; ++i) {
        unsigned char c = (unsigned char)q[i];
        if (c >= ALPHABET_SIZE) { node = 0; continue; }
        int next = d_children[node * ALPHABET_SIZE + c];
        while (next == -1 && node != 0) {
            node = d_fail[node];
            next = d_children[node * ALPHABET_SIZE + c];
        }
        node = (next != -1 ? next : 0);

        // collect new patterns
        uint64_t newL = d_maskLow[node] & ~maskL;
        while (newL) {
            int pid = __ffsll(newL) - 1;
            risk += d_patternWeights[pid];
            newL &= newL - 1;
        }
        maskL |= d_maskLow[node];

        uint64_t newH = d_maskHigh[node] & ~maskH;
        while (newH) {
            int pid = __ffsll(newH) - 1 + 64;
            risk += d_patternWeights[pid];
            newH &= newH - 1;
        }
        maskH |= d_maskHigh[node];
    }

    d_results[tid] = risk;
}

// ------------------------
// Host Aho–Corasick class
// ------------------------
struct TrieNode {
    unordered_map<char, TrieNode*> children;
    TrieNode* fail;
    vector<int> out;  // pattern IDs
    TrieNode(): fail(nullptr) {}
};

class AhoCorasick {
public:
    AhoCorasick() { root = new TrieNode(); nodes.push_back(root); }

    void insert(const string& pat, int id) {
        TrieNode* u = root;
        for (char ch : pat) {
            if (!u->children.count(ch)) {
                TrieNode* v = new TrieNode();
                u->children[ch] = v;
                nodes.push_back(v);
            }
            u = u->children[ch];
        }
        u->out.push_back(id);
    }

    void build() {
        queue<TrieNode*> q;
        root->fail = root;
        for (auto &kv : root->children) {
            kv.second->fail = root;
            q.push(kv.second);
        }
        while (!q.empty()) {
            TrieNode* u = q.front(); q.pop();
            for (auto &kv : u->children) {
                char ch = kv.first;
                TrieNode* v = kv.second;
                TrieNode* f = u->fail;
                while (f != root && !f->children.count(ch)) f = f->fail;
                if (f->children.count(ch)) f = f->children[ch];
                v->fail = f;
                v->out.insert(v->out.end(), f->out.begin(), f->out.end());
                q.push(v);
            }
        }
    }

    int nodeCount()    const { return (int)nodes.size(); }
    int patternCount() const { return (int)patternWeights.size(); }

    TrieNode* getNode(int idx) const { return nodes[idx]; }
    int getFail(int idx) const {
        TrieNode* u = nodes[idx];
        return (u->fail == root ? 0 : (int)(find(nodes.begin(), nodes.end(), u->fail) - nodes.begin()));
    }
    const unordered_map<char, TrieNode*>& getChildren(int idx) const {
        return nodes[idx]->children;
    }
    const vector<int>& getOut(int idx) const { return nodes[idx]->out; }

    void setPatternWeights(const vector<int>& w) { patternWeights = w; }
    const vector<int>& getPatternWeights() const { return patternWeights; }

    // Expose nodes for flattening
    const vector<TrieNode*>& getNodes() const { return nodes; }

private:
    TrieNode* root;
    vector<TrieNode*> nodes;
    vector<int> patternWeights;
};

// Normalize to lowercase
string normalize(const string &s) {
    string r = s;
    transform(r.begin(), r.end(), r.begin(), ::tolower);
    return r;
}

// Host classification
string classifyRisk(int score) {
    if (score <= 30)   return "low";
    if (score <= 70)   return "medium";
    if (score <= 90)   return "high";
    return "critical";
}

// Flatten automaton into host arrays
void flattenAutomaton(
    AhoCorasick& ac,
    vector<int>&      h_children,
    vector<int>&      h_fail,
    vector<uint64_t>& h_maskLow,
    vector<uint64_t>& h_maskHigh)
{
    int N = ac.nodeCount();
    h_children.assign(N * ALPHABET_SIZE, -1);
    h_fail   .assign(N, 0);
    h_maskLow.assign(N, 0ULL);
    h_maskHigh.assign(N, 0ULL);
    const auto& nodes = ac.getNodes();

    for (int u = 0; u < N; ++u) {
        h_fail[u] = ac.getFail(u);
        for (auto &kv : ac.getChildren(u)) {
            unsigned char c = kv.first;
            auto it = find(nodes.begin(), nodes.end(), kv.second);
            int v = (int)(it - nodes.begin());
            h_children[u * ALPHABET_SIZE + c] = v;
        }
        for (int pid : ac.getOut(u)) {
            if (pid < 64)
                h_maskLow[u] |= (1ULL << pid);
            else
                h_maskHigh[u] |= (1ULL << (pid - 64));
        }
    }
}

int main() {

    ofstream out("results.txt");
    streambuf* coutbuf = cout.rdbuf(); // save old buf
    cout.rdbuf(out.rdbuf());

    // 1) Define and weight patterns (expanded list)
    vector<string> rawPatterns = {
        "' or", "\" or", "' ||", "\" ||", "= or", "= ||", "' =", "' >=", "' <=", "' <>",
        "\" =", "\" !=", "= =", "= <", " >=", " <=", "' union", "' select", "' from",
        "union select", "select from", "' convert(", "' avg(", "' round(", "' sum(", "' max(", "' min(",
        ") convert(", ") avg(", ") round(", ") sum(", ") max(", ") min(", "' delete", "' drop",
        "' insert", "' truncate", "' update", "' alter", ", delete", "; drop", "; insert", "; delete", ", drop", "; truncate", "; exec", "xp_cmdshell",
        "; truncate", "' ; update", "like or", "like ||", "' %", "like %", " %", "</script>", "</script >",
        "union", "select", "drop", "insert", "delete", "update", "or 1=1", "--", "#", "/*", "*/",
        "sleep(", "benchmark(", "count(*)", "information_schema.schemata", "null", "version(", "current_user",
        "outfile", "load_file"
    };
    int P = rawPatterns.size();
    vector<string> patterns(P);
    vector<int> weights(P);
    for (int i = 0; i < P; ++i) {
        patterns[i] = normalize(rawPatterns[i]);
        const auto &pat = patterns[i];
        if (pat.find("; drop")!=string::npos || pat.find("xp_cmdshell")!=string::npos ||
            pat.find("; exec")!=string::npos || pat.find("outfile")!=string::npos ||
            pat.find("load_file")!=string::npos)
            weights[i] = 100;
        else if (pat.find("; delete")!=string::npos || pat.find("; insert")!=string::npos ||
                 pat.find("; truncate")!=string::npos || pat.find("; update")!=string::npos ||
                 pat.find("sleep(")!=string::npos || pat.find("version(")!=string::npos ||
                 pat.find("current_user")!=string::npos)
            weights[i] = 15;
        else
            weights[i] = 10;
    }

    // 2) Build host automaton
    AhoCorasick ac;
    ac.setPatternWeights(weights);
    for (int i = 0; i < P; ++i) ac.insert(patterns[i], i);
    ac.build();

    // 3) Flatten and copy to device
    vector<int>      h_children;
    vector<int>      h_fail;
    vector<uint64_t> h_maskLow, h_maskHigh;
    flattenAutomaton(ac, h_children, h_fail, h_maskLow, h_maskHigh);
    int N = ac.nodeCount();
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_children), h_children.data(), N * ALPHABET_SIZE * sizeof(int)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_fail),     h_fail.data(),     N * sizeof(int)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_maskLow),  h_maskLow.data(),  N * sizeof(uint64_t)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_maskHigh), h_maskHigh.data(), N * sizeof(uint64_t)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_patternWeights), weights.data(), P * sizeof(int)));

    // 4) Read queries and expected risks
    ifstream infile("sqli_dataset_Low_New.csv");
    if (!infile.is_open()) {
        cerr << "Error: could not open CSV file.\n";
        return EXIT_FAILURE;
    }
    string line;
    vector<string> queries;
    vector<string> expected;
    // skip header
    getline(infile, line);
    while (getline(infile, line)) {
        if (line.empty()) continue;
        stringstream ss(line);
        string q, expRisk, expScore;
        getline(ss, q, ',');
        getline(ss, expRisk, ',');
        getline(ss, expScore, ',');
        queries.push_back(normalize(q));
        expected.push_back(expRisk);
    }
    infile.close();
    int Q = queries.size();

    // build flat buffer
    vector<int>  h_offsets(Q), h_lengths(Q);
    int totalLen = 0;
    for (int i = 0; i < Q; ++i) {
        h_offsets[i] = totalLen;
        h_lengths[i] = queries[i].size();
        totalLen += h_lengths[i];
    }
    vector<char> h_buffer(totalLen);
    for (int i = 0; i < Q; ++i)
        memcpy(&h_buffer[h_offsets[i]], queries[i].data(), h_lengths[i]);

    // 5) Allocate device memory
    char* d_queries;   int* d_offsets;   int* d_lengths;   int* d_results;
    CUDA_CHECK(hipMalloc(&d_queries, totalLen * sizeof(char)));
    CUDA_CHECK(hipMalloc(&d_offsets, Q * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_lengths, Q * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_results, Q * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_queries, h_buffer.data(), totalLen * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_offsets, h_offsets.data(), Q * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_lengths, h_lengths.data(), Q * sizeof(int), hipMemcpyHostToDevice));

    // 6) Launch kernel
    int threads = 256;
    int blocks  = (Q + threads - 1) / threads;
    ahoSearchKernel<<<blocks, threads>>>(d_queries, d_offsets, d_lengths, Q, d_results);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // 7) Copy back, compare, and compute accuracy
    vector<int> h_results(Q);
    CUDA_CHECK(hipMemcpy(h_results.data(), d_results, Q * sizeof(int), hipMemcpyDeviceToHost));

    int correct = 0;
    for (int i = 0; i < Q; ++i) {
        string compRisk = classifyRisk(h_results[i]);
        bool match = (compRisk == expected[i]);
        if (match) ++correct;
        cout << "Query " << i << ": computed=" << compRisk
                  << ", expected=" << expected[i]
                  << (match ? " [OK]" : " [Mismatch]") << "\n";
    }
    double accuracy = Q ? (100.0 * correct / Q) : 0.0;
    cout << "\nTotal: " << Q << ", Correct: " << correct
              << ", Accuracy: " << accuracy << "%\n";

    // cleanup
    CUDA_CHECK(hipFree(d_queries));
    CUDA_CHECK(hipFree(d_offsets));
    CUDA_CHECK(hipFree(d_lengths));
    CUDA_CHECK(hipFree(d_results));

    cout.rdbuf(coutbuf); // restore old buf if you want to print to console again

    return 0;
}
